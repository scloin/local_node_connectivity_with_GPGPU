#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2022, NVIDIA Corporation
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "gpu_graph.hpp"
#include "cuda_helper.hpp"

constexpr int n_kernel = 2;
constexpr int n_iteration = 40000;

// __global__ void shortKernel(float *out_d, const float *in_d, int N, float f){
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   if(idx < N) { 
//       out_d[idx] = f * in_d[idx];
//   }
// }

// __global__ void initKernel(float *ptr, int N, float f){
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   if(idx < N) { 
//       ptr[idx] = f;
//   }
// }

void run_kernels_graph(float *out_d, float *in_d, int size, float f, gpu_graph_t &g, hipStream_t s)
{
  constexpr int threads = 256;
  int blocks = (size + threads - 1) / threads;

  for(int i = 0; i < n_kernel; i++){
    hipKernelNodeParams params;
    params.blockDim = {static_cast<unsigned int>(threads), 1, 1};
    params.gridDim = {static_cast<unsigned int>(blocks), 1, 1};
    params.sharedMemBytes = 0;
    params.func = reinterpret_cast<void *>(shortKernel);
    void *args[] = {&out_d, &in_d, &size, &f};
    params.kernelParams = args;
    params.extra = nullptr;

    if (g.state() == gpu_graph_t::state_t::capture) {
      // Static kernels
      shortKernel<<<blocks, threads, 0, s>>>(out_d, in_d, size, 1.004f);
      shortKernel<<<blocks, threads, 0, s>>>(in_d, out_d, size, 1.004f);

      // kernels with dynamic parameter `f`
      g.add_kernel_node(i * 2 + 0, params, s);
      params.kernelParams[0] = &in_d;
      params.kernelParams[1] = &out_d;
      g.add_kernel_node(i * 2 + 1, params, s);
    } else if (g.state() == gpu_graph_t::state_t::update) {
      // Update the kernel nodes
      g.update_kernel_node(i * 2 + 0, params);
      params.kernelParams[0] = &in_d;
      params.kernelParams[1] = &out_d;
      g.update_kernel_node(i * 2 + 1, params);
    }
  } 
}

int main() 
{
  gpu_graph_t _graph;

  cudaErrCheck(hipMalloc(&out_d, bytes));
  cudaErrCheck(hipMalloc(&in_d, bytes));

  hipStream_t stream;
  cudaErrCheck(hipStreamCreate(&stream));

  
  auto wrap_obj_graph = [&](gpu_graph_t &g, hipStream_t s) {
    run_kernels_graph(out_d, in_d, size, scale, g, s);
  };

  for(int i = 0; i < n_iteration; i++){
    scale = 1.0f + i * 0.001f;
    _graph.wrap(wrap_obj_graph, stream);
  }
  
  // Finalize memory, stream, events
  cudaErrCheck(hipStreamDestroy(stream));
  // cudaErrCheck(hipEventDestroy(start));
  // cudaErrCheck(hipEventDestroy(stop));

}
