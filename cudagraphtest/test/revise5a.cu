#include "hip/hip_runtime.h"
/*
add POOL
merge memcpy
[X] bfsmain으로 loop end check를 kernel로
loopend 를 kernel로
graph 적용 > done!
overwrap test..
*/

#include "revise5a.hpp"

#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 32


using namespace std;

int main(){

    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 
    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen); 

    gpu_graph_t _graph;
    gpu_graph_t _graph1;

    int *h_data = (int*)malloc((elen+dlen)*sizeof(int)); 
    h_edges= h_data;
    h_dest= &h_data[elen];
    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);

    int* devmem;
    CUDA_CHECK(hipMalloc((void**)&devmem, (elen+dlen+6*(elen-1)+3)*sizeof(int)));

    int* devmem1;
    CUDA_CHECK(hipMalloc((void**)&devmem1, (elen+dlen+6*(elen-1)+3)*sizeof(int)));

    pool P=init_all(elen, dlen, devmem);
    pool P1=init_all(elen, dlen, devmem1);

    CUDA_CHECK(hipMemcpy(P1.d_edges, h_edges, (dlen+elen)*sizeof(int), hipMemcpyHostToDevice)); 

    P.d_edges = P1.d_edges;
    P.d_dest = P1.d_dest;

    int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 

    FILE* fp1 = fopen("result/gtest.txt","w"); 

    auto wrap_obj_graph = [&](gpu_graph_t &g, hipStream_t s) {
        pool C = P;
        run_kernels_graph(C.d_p_frontier, C.d_p_frontier_tail, C.d_c_frontier, C.d_c_frontier_tail,
            C.d_edges, C.d_dest, C.d_label,C.d_visited,
            C.numVertex, C.source, C.target, C.h_label, C.check, g, s,C.state);
    };

    auto wrap_obj_graph1 = [&](gpu_graph_t &g, hipStream_t s) {
        pool C = P1;
        run_kernels_graph(C.d_p_frontier, C.d_p_frontier_tail, C.d_c_frontier, C.d_c_frontier_tail,
            C.d_edges, C.d_dest, C.d_label,C.d_visited,
            C.numVertex, C.source, C.target, C.h_label, C.check, g, s,C.state);
    };

    int* Ad;

    while(P.source<P.numVertex){
        P.target=P.source+1;
        P1.source=P.source;
        P1.target=P.target;

    while(P.target<P.numVertex){
    for (int i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
    }
    int count=0; 

    int N = degree(h_dest,h_edges,P.source);

    Ad=&(h_dest[h_edges[P.source]]);

    int check=0;
    for(int j = 0; j<N;j++){
        if(*(Ad+j)==P.target){
            check=1; 

            break;
        }
    }
    if(check){
        P.h_visited[(P.source<<1)+1]=-1;
        P.h_visited[(P.target<<1)]=-1;
        count++;
    }
    //thread t1;
    thread t2;
    while(1){ 
    BFS_host(P, &wrap_obj_graph, &_graph);

    //thread test.. 
    t2=thread{degree,h_dest,h_edges,P.source};
    //


    //t1.join(); 
    t2.join();
    list<int> list1;
    int num = *P.h_returned;
    if(num==-1) {
        break;
        }
    count++;
    list<int>::iterator begin_iter = list1.begin();
    list<int>::iterator end_iter = list1.end();
    list1.insert(end_iter, num);
    begin_iter--; 

    int i=num; 

    for (int KK=0; KK<P.numVertex;KK++){
            exclude_S[KK] = P.h_label[KK*2];
            exclude_T[KK] = P.h_label[KK*2+1];
        } 

    while((i>-1)){
        if(i!=num) {
        list1.insert(end_iter, i);}

        i = path(exclude_T,i,h_dest,h_edges,0,elen-1);
        } 

    i=num;
    while((i>-1)){
        if(i!=num) {
        list1.insert(begin_iter, i);
        begin_iter--;}
        i = path(exclude_S,i,h_dest,h_edges,1,elen-1);
        }
    while (list1.empty()==0) {
        i=list1.front();
        P.h_visited[i<<1]=-1;
        P.h_visited[(i<<1)+1]=-1;
        list1.pop_front(); 

    } 

    list1.clear();

    }
    
    
    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    P.target++;

    } 

    P.source++;}
    fclose(fp1); 
    
    _graph.dest_graph();
    //free

    CUDA_CHECK(hipFree(devmem));
    CUDA_CHECK(hipFree(devmem1));
    free(h_data);

    free(P.h_label);
    free(P1.h_label);

    free(P.h_returned);
    free(P1.h_returned);
    free(exclude_T);
    free(exclude_S);
    CUDA_CHECK( hipStreamDestroy(P.stream0));
    CUDA_CHECK( hipStreamDestroy(P1.stream0));
    hipDeviceReset();
    return 0;
} 


template<class Obj>
void BFS_host(pool P, Obj* wrap_obj_graph,gpu_graph_t* _graph)
{

    int* h_p_frontier_tail=&P.h_label[4*P.numVertex];
    *h_p_frontier_tail = 2;

    int S = P.source<<1;
    int T = (P.target<<1)+1; 

    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    
    for (int i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i]=0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpy(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice)); 
    memset_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream0>>>(P.d_label, P.d_visited, P.d_frontier, S, T, P.numVertex,P.d_p_frontier_tail, P.check);

    *P.h_check=0; 

    while (*h_p_frontier_tail > 0&&*P.h_check==0) { 
        (*_graph).wrap(*wrap_obj_graph, P.stream0);
        hipDeviceSynchronize();
    }
    int min = P.numVertex;
    int meet = -1; 
    *P.state=2;
    for(int k=0;k<P.numVertex;k++){
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&k!=P.source&&k!=P.target){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]&&(P.h_label[2*k]*P.h_label[2*k+1])>0){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }
    //printf("====>>%d\n", min);
    *P.h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    //__syncthreads();
        if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail, int * check){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        //*(d_visited+tid) = 0;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }
    *check=0;

}

__global__ void checking(int num,int* visited, int source, int target, int* check,int *d_c_frontier,int*d_p_frontier){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid<num)
        if(visited[2*tid]==1&&visited[2*tid+1]==1&&tid!=source&&tid!=target
){

            *check=1;
        }

}

/* 
1. 파라미터 변경필요 V
2. memcpy node 추가 V
3. statci kernel 추가 V
>>static도 graph에 포함되는지 확인필요!
>>>실행해보기!!!
*/
void run_kernels_graph(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail,  
    int* edges, int* dest, int* label, int* visited, 
    int numVertex, int source, int target, int* h_label, int* check, gpu_graph_t &g, hipStream_t s, int* state){
    //constexpr int threads = 256;
    //int blocks = (size + threads - 1) / threads;
    int num_blocks = (h_label[4*numVertex]+BLOCK_SIZE-1) / BLOCK_SIZE;
    int num_blocks1 = ((numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
        
    hipKernelNodeParams params;
        params.blockDim = {static_cast<unsigned int>(BLOCK_SIZE), 1, 1};
        params.gridDim = {static_cast<unsigned int>(num_blocks), 1, 1};
        params.sharedMemBytes = 0;
        params.func = reinterpret_cast<void *>(BFS_Bqueue_kernel);
        void *args[] = {&p_frontier, &p_frontier_tail, &c_frontier, &c_frontier_tail, &edges, &dest, &label, &visited};
        params.kernelParams = args;
        params.extra = nullptr;

    hipKernelNodeParams params1;
        params1.blockDim = {static_cast<unsigned int>(BLOCK_SIZE), 1, 1};
        params1.gridDim = {static_cast<unsigned int>(num_blocks1), 1, 1};
        params1.sharedMemBytes = 0;
        params1.func = reinterpret_cast<void *>(checking);
        void *args1[] = {&numVertex, &visited, &source, &target, &check, &c_frontier ,&p_frontier};
        params1.kernelParams = args1;
        params1.extra = nullptr;

    void* temp;
    //if (g.state() == gpu_graph_t::state_t::capture) {
    if (*state==0) {
      g.add_kernel_node(0, params, s);
      g.add_kernel_node(1, params1, s);
      g.add_memcpy_node(2, label, h_label, (4*numVertex+2)*sizeof(int), s);
      *state=1;
    //} else if (g.state() == gpu_graph_t::state_t::update) {
    } else if (*state==1) {
        temp = params.kernelParams[0];
        params.kernelParams[0]= params.kernelParams[2];
        params.kernelParams[2]=temp;

      g.update_kernel_node(0, params);
      g.update_kernel_node(1, params1);
      *state=2;
    }
    else if (*state==2) {
      g.update_kernel_node(0, params);
      g.update_kernel_node(1, params1);
      *state=1;
    }

}

pool init_all(int elen, int dlen, int* devmem){
    pool P;
    CUDA_CHECK( hipStreamCreate(&P.stream0) );
    P.numVertex=elen-1;
    P.h_label=(int*)malloc((4*(P.numVertex)+2)*sizeof(int)); 
    P.h_visited=&(P.h_label[2*P.numVertex]);
    P.h_check  =&(P.h_label[4*P.numVertex+1]);
    P.state=(int*)malloc(sizeof(int)); 
    *P.state=0;

    P.d_edges=devmem;
    P.d_dest=&(devmem[elen]);
    P.d_label=&(devmem[dlen+elen]);
    P.d_visited=&(P.d_label[2*P.numVertex]);

    P.d_p_frontier_tail=&(P.d_label[4*P.numVertex]);
    P.check=&(P.d_label[4*P.numVertex+1]);

    P.d_frontier=&(P.d_label[4*P.numVertex+2]);
    P.d_c_frontier_tail=&(P.d_label[6*P.numVertex+2]);

    P.d_c_frontier = &P.d_frontier[0];
    P.d_p_frontier = &P.d_frontier[P.numVertex];

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 

    return P;
}