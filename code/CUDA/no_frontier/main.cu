#include <revise5t.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 512

using namespace std;

int main(){

    ///////////////////////////////////////////////////////
    /*read file*/
    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 
    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen);
    int *h_data;
    h_data = (int*)malloc((elen+dlen)*sizeof(int)); 
    //CUDA_CHECK(hipHostMalloc((void**)&h_data, (elen+dlen)*sizeof(int)));
    h_edges= h_data;
    h_dest= &h_data[elen];
    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);

    ///////////////////////////////////////////////////////
    /*alloc & init*/
    int* devmem; 
    int* devmem1; 
    int* devmem2;
    int* devmem3;
    int* d_edges;
    CUDA_CHECK(hipMalloc((void**)&d_edges, (elen+dlen)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&devmem, (6*(elen-1)+2)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&devmem1, (6*(elen-1)+2)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&devmem2, (6*(elen-1)+2)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&devmem3, (6*(elen-1)+2)*sizeof(int)));

    pool P0 = init_pool(elen, dlen, devmem, d_edges);
    pool P1 = init_pool(elen, dlen, devmem1, d_edges);
    pool P2 = init_pool(elen, dlen, devmem2, d_edges);
    pool P3 = init_pool(elen, dlen, devmem3, d_edges);
    CUDA_CHECK(hipMemcpy(d_edges, h_edges, (dlen+elen)*sizeof(int), hipMemcpyHostToDevice)); 
    thread t0, t1, t2, t3;
    FILE* fp1 = fopen("result/no_frontier.txt","w"); 

    ///////////////////////////////////////////////////////
    /*compute*/
    P0.target=P0.source+1;
    P1.target=P0.source+2;
    P2.target=P0.source+3;
    P3.target=P0.source+4;
    P1.source=P0.source; 
    P2.source=P0.source;
    t0=thread{compute,h_dest,h_edges,P0,fp1};
    t1=thread{compute,h_dest,h_edges,P1,fp1};
    t2=thread{compute,h_dest,h_edges,P2,fp1};
    t3=thread{compute,h_dest,h_edges,P3,fp1};
    //compute(h_dest,h_edges,P0,fp1);
    t0.join();
    t1.join();
    t2.join();
    t3.join();

    fclose(fp1);

    ///////////////////////////////////////////////////////
    /*free*/
    CUDA_CHECK(hipFree(devmem));
    CUDA_CHECK(hipFree(devmem1));
    CUDA_CHECK(hipFree(devmem2));
    CUDA_CHECK(hipFree(devmem3));
    CUDA_CHECK(hipFree(d_edges)); 
    CUDA_CHECK( hipStreamDestroy(P0.stream));
    CUDA_CHECK( hipStreamDestroy(P1.stream));
    CUDA_CHECK( hipStreamDestroy(P2.stream));
    CUDA_CHECK( hipStreamDestroy(P3.stream));
    free(h_data);
    free(P0.h_label);
    free(P0.h_returned);
    free(P1.h_label);
    free(P1.h_returned);
    free(P2.h_label);
    free(P2.h_returned);
    free(P3.h_label);
    free(P3.h_returned);
    //hipHostFree(h_data);
    hipDeviceReset();
    return 0;
} 