#include "hip/hip_runtime.h"
#include <revise4t.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 512

void BFS_host(pool P)
{
    int* h_p_frontier_tail=&P.h_visited[2*P.numVertex];
    *h_p_frontier_tail = 2;

    int S = P.source<<1;
    int T = (P.target<<1)+1; 
    int i; int k; int * temp;
    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(P.d_label, P.d_visited, P.d_frontier, S, T, P.numVertex,P.d_p_frontier_tail);
    for (i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpyAsync(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice,P.stream)); 

    int *d_c_frontier = &P.d_frontier[0];
    int *d_p_frontier = &P.d_frontier[P.numVertex];
    int check=0; 
    
    while (*h_p_frontier_tail > 0&&check==0) { 

        num_blocks = (*h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
        BFS_Bqueue_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        //BFS_Bqueue<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        
        //CUDA_CHECK(hipMemcpy(P.h_visited, P.d_visited, (2*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpyAsync(P.h_label, P.d_label, (4*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost,P.stream)); 

        for(k=0;k<P.numVertex;k++){
            if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&k!=P.source&&k!=P.target){
                check=1;
                break;
            }
        } 
        temp = d_c_frontier;
        d_c_frontier = d_p_frontier;
        d_p_frontier = temp;


    }
    //CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, 2*P.numVertex*sizeof(int), hipMemcpyDeviceToHost)); 

    int min = P.numVertex;
    int meet = -1; 
    //printf("\n...\n");
    for(k=0;k<P.numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&P.h_label[2*k]*P.h_label[2*k+1]!=0){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }
    //if(P.source==0 && P.target==282) printf("meet : %d, visited : [%d,%d], label : [%d, %d]\n", meet, P.h_visited[2*meet], P.h_visited[2*meet+1], P.h_label[2*meet], P.h_label[2*meet+1]);
    *P.h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 
    __shared__ int w_queue[16][16], our_c_frontier_s_tail[16];
    __shared__ int w_tail[16];

    if (threadIdx.x <16) {
        c_frontier_tail_s = 0;
        w_tail[threadIdx.x]=0;
    }
    __syncthreads(); 
    const int wid = threadIdx.x%16;
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (was_visited==0) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                //const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                const int my_w_tail = atomicAdd(&w_tail[wid], 1);
                if (my_w_tail < 16) {
                    w_queue[wid][my_w_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    w_tail[wid]=16;
                    const int my_block_tail = atomicAdd(&c_frontier_tail_s, 1);
                    if (my_block_tail < BLOCK_QUEUE_SIZE) {
                        c_frontier_s[my_block_tail] = (dest[i]<<1)+my_state;
                    }
                    else {
                        c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                        const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                        c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                    }
                    //c_frontier_s[my_block_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x <16) {
        our_c_frontier_s_tail[wid] = atomicAdd(&c_frontier_tail_s, w_tail[wid]);
        //printf("our_c_- %d, w_tail%d\n", our_c_frontier_s_tail[wid], w_tail[wid]);
    }
    __syncthreads(); 

    for (int i = threadIdx.x/16; i < w_tail[wid]; i += 16) {
        //printf("tid=%d, wqueue[%d][%d]=%d, c_frontier_s[%d]\n", threadIdx.x,wid,i,w_queue[wid][i],our_c_frontier_s_tail[wid] + i);
        if(our_c_frontier_s_tail[wid] + i>=BLOCK_QUEUE_SIZE) {
            const int second_global_tail = atomicAdd(c_frontier_tail, 1);
            c_frontier[second_global_tail] = w_queue[wid][i];
            }
        else{
            c_frontier_s[our_c_frontier_s_tail[wid] + i] = w_queue[wid][i];
            }    
        }
    __syncthreads(); 


    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    __syncthreads();
        if (tid == 0) {

        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }

}

pool init_pool(int elen, int dlen, int* devmem, int* d_edges){

    pool P;
    CUDA_CHECK( hipStreamCreate(&P.stream) );
    P.numVertex=elen-1;

    P.d_edges=d_edges;
    P.d_dest=&d_edges[elen];

    P.h_label=(int*)malloc((4*(P.numVertex)+2)*sizeof(int)); 
    //CUDA_CHECK(hipHostMalloc((void**)&P.h_label, (4*(P.numVertex)+2)*sizeof(int)));
    P.h_visited=&(P.h_label[2*P.numVertex]);

    P.d_label=devmem;
    P.d_visited=&(devmem[2*P.numVertex]);

    P.d_frontier=&(devmem[4*P.numVertex+1]);
    P.d_c_frontier_tail=&(devmem[6*P.numVertex+1]);
    P.d_p_frontier_tail=&(devmem[4*P.numVertex]);

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 

    return P;
}

void compute(int* h_dest,int * h_edges, pool P,FILE* fp1){
    int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 
    int init = P.target; 
<<<<<<< HEAD
    int count; int N; int* Ad; int num; int i; int j; int KK;
    while(P.source<P.numVertex){
        P.target=P.source+init;
        while(P.target<P.numVertex){
    count=0; 
=======
    while(P.source<P.numVertex){
        P.target=P.source+init;
        while(P.target<P.numVertex){
    int count=0; 
>>>>>>> beeae6054c79a5b3d5057253c7de5bd990b5e31f
    //verify that it is connected directly
    N = degree(h_dest,h_edges,P.source);
    Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    //int check=0;
    for(j = 0; j<N;j++){
        if(Ad[j]==P.target){
            P.h_visited[(P.source<<1)+1]=-1;
            P.h_visited[(P.target<<1)]=-1;
            count++; 
            break;
        }
    }
    free(Ad);
    //loop while count every component
    while(1){ 
    
<<<<<<< HEAD
        //BFS_host(P);
        BFS_host2(P);
        num = *P.h_returned;
=======
        BFS_host(P);
        //std::list<int> list;//
        int num = *P.h_returned;
>>>>>>> beeae6054c79a5b3d5057253c7de5bd990b5e31f
        if(num==-1) {
            break;
            }
        count++;
<<<<<<< HEAD

        i=num;
        j=num; 
=======
        //std::list<int>::iterator begin_iter = list.begin();//
        //std::list<int>::iterator end_iter = list.end();//
        //list.insert(end_iter, num);//
        //begin_iter--; //

        int i=num; 
>>>>>>> beeae6054c79a5b3d5057253c7de5bd990b5e31f
        P.h_visited[i<<1]=-1;
        P.h_visited[(i<<1)+1]=-1;

        for (KK=0; KK<P.numVertex;KK++){
                exclude_S[KK] = P.h_label[KK*2];
                exclude_T[KK] = P.h_label[KK*2+1];
            } 

<<<<<<< HEAD
        int tempi=0;
        int tempj=0;
        int error =0;
        while((i>-1)||(j>-1)){
            if(i>-1){
                if(i!=num) {
                P.h_visited[i<<1]=-1;
                P.h_visited[(i<<1)+1]=-1;
                }
                tempi=i;
                i = path(exclude_T,i,h_dest,h_edges,0,P.numVertex);
                if(tempi==i){
                    error=1;
                    break;
                }
            }
            if(j>-1){
                if(j!=num) {
                P.h_visited[j<<1]=-1;
                P.h_visited[(j<<1)+1]=-1;
                }
                tempj=j;
                j = path(exclude_S,j,h_dest,h_edges,1,P.numVertex);
                if(tempj==j){
                    error=1;
                    break;
                }
            }
        }
        if(error==1){
            count--;
            break;
        }
    }
    //record count
    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    
    P.target+=4;
    //P.target++;
    for (i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
        }
    P.source++;
    }

    free(exclude_S);
    free(exclude_T);
}

__global__ void BFS_Bqueue(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    //__syncthreads();
        if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 
=======
        while((i>-1)){
            if(i!=num) {
            P.h_visited[i<<1]=-1;
            P.h_visited[(i<<1)+1]=-1;
            }
            i = path(exclude_T,i,h_dest,h_edges,0,P.numVertex);
            } 
        i=num;
        while((i>-1)){
            if(i!=num) {
            P.h_visited[i<<1]=-1;
            P.h_visited[(i<<1)+1]=-1;
            }
            i = path(exclude_S,i,h_dest,h_edges,1,P.numVertex); 
>>>>>>> beeae6054c79a5b3d5057253c7de5bd990b5e31f

/*
BFS with Cuda c++, using shared memory, and warp level queue with less atomic operation
*/
__global__ void BFS_less_atomic(int* d_edges, int* d_dest, int* d_label, int* d_visited, int numVertex, int level){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    const int n = 2*numVertex;
    if (tid < n) {
        const int my_vertex = tid>>1;
        const int my_state = tid&1;
        if (d_label[tid] == level) {
            for (int i = d_edges[my_vertex]; i < d_edges[my_vertex+1]; i++) {
                int v = (d_dest[i]<<1)+my_state;
                if (d_visited[v] == 0) {
                    d_label[v] = level + 1;
                    d_visited[v] = 1;
                }
            }
        }
    }
}

/*
Revise of BFS_host:
    - delete the frontier queue
    - use BFS_less_atomic() to replace BFS_Bqueue_kernel()
*/
void BFS_host2(pool P)
{
    int S = P.source<<1;
    int T = (P.target<<1)+1; 
    int i; int k;
    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel2<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(P.d_label, P.d_visited, S, T, P.numVertex);

    //init and copy visited
    for (i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpyAsync(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice,P.stream)); 

    int check=0; 
    int level = 0;

    num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    int csum =0;
    while (check==0) { 
        //hipMemcpyAsync(d_done, &true_value, sizeof(int), hipMemcpyHostToDevice,P.stream);
        //CUDA_CHECK(hipMemsetAsync(d_done, 0, sizeof(int),P.stream));

        BFS_less_atomic<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(P.d_edges, P.d_dest, P.d_label, P.d_visited, P.numVertex, level);
        hipDeviceSynchronize();
        CUDA_CHECK(hipMemcpyAsync(P.h_label, P.d_label, (4*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost,P.stream)); 
        
        hipDeviceSynchronize();
        for(k=0;k<P.numVertex;k++){
            if ((P.h_label[2*k]==level)||(P.h_label[2*k+1]==level)){
                csum++;
            };
            if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&k!=P.source&&k!=P.target){
                check=1;
                break;
            }
<<<<<<< HEAD
        } 
        if(csum==0){
            check=1;
        }
        csum=0;
        level++;
    }

    int min = P.numVertex;
    int meet = -1; 
    for(k=0;k<P.numVertex;k++){
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&P.h_label[2*k]*P.h_label[2*k+1]!=0){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }

    *P.h_returned=meet;
} 

__global__ void memset_kernel2(int* d_label, int* d_visited, int S, int T, int NUM){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
    }

=======
    }
    //record count
    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    
    P.target+=3;
    for (int i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
        }
    P.source++;
    }

    free(exclude_S);
    free(exclude_T);
>>>>>>> beeae6054c79a5b3d5057253c7de5bd990b5e31f
}