#include "hip/hip_runtime.h"
#include <revise4t.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 32 

void BFS_host(pool P)
{
    int* h_p_frontier_tail=&P.h_visited[2*P.numVertex];
    *h_p_frontier_tail = 2;

    int S = P.source<<1;
    int T = (P.target<<1)+1; 

    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(P.d_label, P.d_visited, P.d_frontier, S, T, P.numVertex,P.d_p_frontier_tail);
    for (int i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpyAsync(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice,P.stream)); 

    int *d_c_frontier = &P.d_frontier[0];
    int *d_p_frontier = &P.d_frontier[P.numVertex];
    int check=0; 

    while (*h_p_frontier_tail > 0&&check==0) { 

        num_blocks = (*h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
        BFS_Bqueue_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        
        //CUDA_CHECK(hipMemcpy(P.h_visited, P.d_visited, (2*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpyAsync(P.h_label, P.d_label, (4*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost,P.stream)); 

        for(int k=0;k<P.numVertex;k++){
            if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&k!=P.source&&k!=P.target){
                check=1;
                break;
            }
        } 

        int* temp = d_c_frontier;
        d_c_frontier = d_p_frontier;
        d_p_frontier = temp;


    }
    //CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, 2*P.numVertex*sizeof(int), hipMemcpyDeviceToHost)); 

    int min = P.numVertex;
    int meet = -1; 

    for(int k=0;k<P.numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }
    *P.h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    //__syncthreads();
        if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }

}

pool init_pool(int elen, int dlen, int* devmem, int* d_edges){

    pool P;
    CUDA_CHECK( hipStreamCreate(&P.stream) );
    P.numVertex=elen-1;

    P.d_edges=d_edges;
    P.d_dest=&d_edges[elen];

    P.h_label=(int*)malloc((4*(P.numVertex)+2)*sizeof(int)); 
    P.h_visited=&(P.h_label[2*P.numVertex]);

    P.d_label=devmem;
    P.d_visited=&(devmem[2*P.numVertex]);

    P.d_frontier=&(devmem[4*P.numVertex+1]);
    P.d_c_frontier_tail=&(devmem[6*P.numVertex+1]);
    P.d_p_frontier_tail=&(devmem[4*P.numVertex]);

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 

    return P;
}

void compute(int* h_dest,int * h_edges, pool P,FILE* fp1){
    int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 
    int init = P.target; 
    while(P.source<P.numVertex){
        P.target=P.source+init;
        while(P.target<P.numVertex){
    int count=0; 
    //verify that it is connected directly
    int N = degree(h_dest,h_edges,P.source);
    int* Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    //int check=0;
    for(int j = 0; j<N;j++){
        if(Ad[j]==P.target){
            P.h_visited[(P.source<<1)+1]=-1;
            P.h_visited[(P.target<<1)]=-1;
            count++; 
            break;
        }
    }
    free(Ad);
    //loop while count every component
    while(1){ 
    
        BFS_host(P);
        //std::list<int> list;//
        int num = *P.h_returned;
        if(num==-1) {
            break;
            }
        count++;
        //std::list<int>::iterator begin_iter = list.begin();//
        //std::list<int>::iterator end_iter = list.end();//
        //list.insert(end_iter, num);//
        //begin_iter--; //

        int i=num; 
        P.h_visited[i<<1]=-1;
        P.h_visited[(i<<1)+1]=-1;

        for (int KK=0; KK<P.numVertex;KK++){
                exclude_S[KK] = P.h_label[KK*2];
                exclude_T[KK] = P.h_label[KK*2+1];
            } 

        while((i>-1)){
            if(i!=num) {
            P.h_visited[i<<1]=-1;
            P.h_visited[(i<<1)+1]=-1;
            }
            i = path(exclude_T,i,h_dest,h_edges,0,P.numVertex);
            } 
        i=num;
        while((i>-1)){
            if(i!=num) {
            P.h_visited[i<<1]=-1;
            P.h_visited[(i<<1)+1]=-1;
            }
            i = path(exclude_S,i,h_dest,h_edges,1,P.numVertex); 

            }
    }
    //record count
    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    
    P.target+=3;
    for (int i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
        }
    P.source++;
    }

    free(exclude_S);
    free(exclude_T);
}