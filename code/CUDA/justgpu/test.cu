#include <justgpu.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 512

using namespace std;

int main(){

    ///////////////////////////////////////////////////////
    /*read file*/
    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 
    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen);
    int *h_data;
    h_data = (int*)malloc((elen+dlen)*sizeof(int)); 
    //CUDA_CHECK(hipHostMalloc((void**)&h_data, (elen+dlen)*sizeof(int)));
    h_edges= h_data;
    h_dest= &h_data[elen];
    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);

    ///////////////////////////////////////////////////////
    /*alloc & init*/
    int* devmem; 
    int* d_edges;
    CUDA_CHECK(hipMalloc((void**)&d_edges, (elen+dlen)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&devmem, (8*(elen-1)+4)*sizeof(int)));

    pool P0 = init_pool(elen, dlen, devmem, d_edges);
    CUDA_CHECK(hipMemcpy(d_edges, h_edges, (dlen+elen)*sizeof(int), hipMemcpyHostToDevice)); 

    ///////////////////////////////////////////////////////
    /*compute*/
    P0.target=P0.source+1;
    compute_test(h_dest,h_edges,P0);



    ///////////////////////////////////////////////////////
    /*free*/
    CUDA_CHECK(hipFree(devmem));
    CUDA_CHECK(hipFree(d_edges)); 
    CUDA_CHECK( hipStreamDestroy(P0.stream));
    free(h_data);
    CUDA_CHECK(hipHostFree(P0.h_label));
    free(P0.h_returned);

    hipDeviceReset();
    return 0;
} 