#include "hip/hip_runtime.h"
#include <justgpu.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 512

/*
짝 홀 나눠 있는거 풀기
BFS 두번으로 바꾸기
*/


void BFS_host(pool P)
{
    int* h_p_frontier_tail=&P.h_visited[2*P.numVertex];
    *h_p_frontier_tail = 1;
    
    int* h_p_frontier_tail1=&P.h_visited[2*P.numVertex+1];
    *h_p_frontier_tail1 = 1;

    int S = P.source;
    int T = P.target; 
    int i; int k; int * temp;
    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(S, T, P.d_label, P.d_frontier, P.numVertex, P.d_p_frontier_tail, P.d_frontier1, P.d_p_frontier_tail1);

    for (i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }

    P.h_visited[P.source] =1;P.h_visited1[P.target] =1; 

    CUDA_CHECK(hipMemcpyAsync(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice,P.stream)); 

    int *d_c_frontier = &P.d_frontier[0];
    int *d_p_frontier = &P.d_frontier[P.numVertex];
    int *d_c_frontier1 = &P.d_frontier1[0];
    int *d_p_frontier1 = &P.d_frontier1[P.numVertex];

    int check=0;
    int turn=0; //S부터 T부터
    while ((*h_p_frontier_tail + *h_p_frontier_tail1) > 0&&check==0) { 

        
        if(turn%2==0){
            num_blocks = (*h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
            BFS_noqueue<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        }
        else{
            num_blocks = (*h_p_frontier_tail1+BLOCK_SIZE-1) / BLOCK_SIZE;
            BFS_noqueue<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier1, P.d_p_frontier_tail1,d_c_frontier1, P.d_c_frontier_tail1, P.d_edges, P.d_dest, P.d_label1, P.d_visited1);
        }
        CUDA_CHECK(hipMemcpyAsync(P.h_label, P.d_label, (4*P.numVertex+2)*sizeof(int), hipMemcpyDeviceToHost,P.stream)); 
        hipDeviceSynchronize();



        for(k=0;k<P.numVertex;k++){
            if(P.h_visited[k]==1&&P.h_visited1[k]==1&&k!=P.source&&k!=P.target){
                check=1;
                // printf("%d\n", turn);
                // for(k=0;k<P.numVertex;k++){
                //     printf("%2d ", P.h_label[k]);
                // }
                //     printf("\n");
                // for(k=0;k<P.numVertex;k++){
                //     printf("%2d ", P.h_label1[k]);
                // }
                // printf("\n");

                // for(k=0;k<P.numVertex;k++){
                //     printf("%2d ", P.h_visited[k]);
                // }
                //     printf("\n");
                // for(k=0;k<P.numVertex;k++){
                //     printf("%2d ", P.h_visited1[k]);
                // }
                // printf("\n");

                break;
            }
        } 
        if(turn%2==0){
            temp = d_c_frontier;
            d_c_frontier = d_p_frontier;
            d_p_frontier = temp;
        }
        else{
            temp = d_c_frontier1;
            d_c_frontier1 = d_p_frontier1;
            d_p_frontier1 = temp;
        }
        turn++;

    }


    int min = P.numVertex;
    int meet = -1; 
    //printf("\n...\n");
    for(k=0;k<P.numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(P.h_visited[k]==1&&P.h_visited1[k]==1&&P.h_label[k]*P.h_label1[k]!=0){
            if(min>P.h_label[k]+P.h_label1[k]){
                min = P.h_label[k]+P.h_label1[k];
                meet =k;
            }
        }
    }

    *P.h_returned=meet;
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int S, int T, int* d_label,int* d_frontier,int numVertex,int* d_p_frontier_tail, int* d_frontier1, int* d_p_frontier_tail1){

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < numVertex*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0; *(d_frontier1+tid) =0;
        if (tid==S||tid==T+numVertex) {
            *(d_label+tid) = 0;
                }
        if (tid ==numVertex) *(d_frontier+tid) = S;
        if (tid ==numVertex+1) *(d_frontier1+tid-1) = T;
        *(d_p_frontier_tail) =1;
        *(d_p_frontier_tail1) =1;
    }

}

pool init_pool(int elen, int dlen, int* devmem, int* d_edges){

    pool P;
    CUDA_CHECK( hipStreamCreate(&P.stream) );
    P.numVertex=elen-1;

    P.d_edges=d_edges;
    P.d_dest=&d_edges[elen];


    CUDA_CHECK(hipHostMalloc((void**)&P.h_label, (4*(P.numVertex)+2)*sizeof(int)));
    P.h_label1 =&(P.h_label[P.numVertex]);
    P.h_visited=&(P.h_label[2*P.numVertex]);
    P.h_visited1=&(P.h_label[3*P.numVertex]);

    P.d_label=devmem;
    P.d_label1=&(devmem[P.numVertex]);
    P.d_visited=&(devmem[2*P.numVertex]);
    P.d_visited1=&(devmem[3*P.numVertex]);


    P.d_frontier=&(devmem[4*P.numVertex+2]);
    P.d_frontier1=&(devmem[6*P.numVertex+4]);
    P.d_c_frontier_tail=&(devmem[6*P.numVertex+2]);
    P.d_p_frontier_tail=&(devmem[4*P.numVertex]);
    P.d_c_frontier_tail1=&(devmem[6*P.numVertex+3]);
    P.d_p_frontier_tail1=&(devmem[4*P.numVertex+1]);

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 

    return P;
}

void compute(int* h_dest,int * h_edges, pool P,FILE* fp1){
    // int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    // int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 
    int init = P.target; 
    int count; int N; int* Ad; int num; int i; int j; //int KK;
    while(P.source<P.numVertex){
        P.target=P.source+init;
        while(P.target<P.numVertex){
    count=0; 
    //verify that it is connected directly
    N = degree(h_dest,h_edges,P.source);
    Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    //int check=0;
    for(j = 0; j<N;j++){
        if(Ad[j]==P.target){
            P.h_visited1[P.source]=-1;
            P.h_visited[P.target]=-1;
            count++; 
            break;
        }
    }
    free(Ad);
    //loop while count every component
    while(1){ 
    
        //BFS_host(P);
        BFS_host(P);
        num = *P.h_returned;

        if(num==-1) {
            break;
            }
        count++;

        i=num;
        j=num; 
        P.h_visited[i]=-1;
        P.h_visited1[i]=-1;

        int tempi=0;
        int tempj=0;
        int error =0;
        while((i>-1)||(j>-1)){
            if(i>-1){
                if(i!=num) {
                P.h_visited[i]=-1;
                P.h_visited1[i]=-1;
                }
                tempi=i;
                i = path(P.h_label1,i,h_dest,h_edges,0,P.numVertex);
                if(tempi==i){
                    error=1;
                    break;
                }
            }
            if(j>-1){
                if(j!=num) {
                P.h_visited[j]=-1;
                P.h_visited1[j]=-1;
                }
                tempj=j;
                j = path(P.h_label,j,h_dest,h_edges,1,P.numVertex);
                if(tempj==j){
                    error=1;
                    break;
                }
            }
        }
        if(error==1){
            count--;
            break;
        }
    }
    //record count
    //fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    
    P.target+=4;
    //P.target++;
    for (i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
        }
    P.source++;
    }

    // free(exclude_S);
    // free(exclude_T);
}

__global__ void BFS_noqueue(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid];

        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[dest[i]]!=-1)
                was_visited = atomicExch(&(visited[dest[i]]), 1);
            if (!was_visited) {
                label[dest[i]] = label[my_vertex] + 1;
                const int my_tail = atomicAdd(c_frontier_tail, 1);
                c_frontier[my_tail] = (dest[i]);
            }
        }
    }
    __syncthreads();
    if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);

    }
} 

void compute_test(int* h_dest,int * h_edges, pool P){
    int count; int N; int* Ad; int num; int i; int j;

    count=0; 
    //verify that it is connected directly
    N = degree(h_dest,h_edges,P.source);
    Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    //int check=0;
    for(j = 0; j<N;j++){
        if(Ad[j]==P.target){
            P.h_visited[P.source]=-1;
            P.h_visited1[P.target]=-1;
            count++; 
        }
    }
    free(Ad);
    //loop while count every component
    while(1){ 
    
        //BFS_host(P);
        BFS_host(P);
        num = *P.h_returned;
        if(num==-1) {
            break;
            }
        count++;

        i=num;
        j=num; 
        P.h_visited[i]=-1;
        P.h_visited1[i]=-1;


        while((i>-1)||(j>-1)){
            if(i>-1){
                if(i!=num) {
                P.h_visited[i]=-1;
                P.h_visited1[i]=-1;
                }
                // tempi=i;
                i = path(P.h_label1,i,h_dest,h_edges,0,P.numVertex);
                // if(tempi==i){
                //     error=1;
                //     break;
                // }
            }
            if(j>-1){
                if(j!=num) {
                P.h_visited[j]=-1;
                P.h_visited[j]=-1;
                }
                // tempj=j;
                j = path(P.h_label,j,h_dest,h_edges,1,P.numVertex);
                // if(tempj==j){
                //     error=1;
                //     break;
                // }
            }
        }
        // if(error==1){
        //     count--;
        //     break;
        // }
    }
    //record count
    //fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    //printf("[%d, %d] %d\n", P.source, P.target, count);
    
}
