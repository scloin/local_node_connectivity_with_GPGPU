#include <revise4t.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 32 

using namespace std;

int main(){

    ///////////////////////////////////////////////////////
    /*read file*/
    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 
    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen); 
    printf("%d\n",elen);
    int *h_data = (int*)malloc((elen+dlen)*sizeof(int)); 
    h_edges= h_data;
    h_dest= &h_data[elen];
    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);

    ///////////////////////////////////////////////////////
    /*alloc & init*/
    int* devmem; int* devmem1;

    CUDA_CHECK(hipMalloc((void**)&devmem, (elen+dlen+6*(elen-1)+2)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&devmem1, (elen+dlen+6*(elen-1)+2)*sizeof(int)));

    pool P0 = init_pool(elen, dlen, devmem);
    pool P1 = init_pool(elen, dlen, devmem1);
    CUDA_CHECK(hipMemcpy(P0.d_edges, h_edges, (dlen+elen)*sizeof(int), hipMemcpyHostToDevice)); 
    P1.d_edges = P0.d_edges; P1.d_dest = P0.d_dest;

    FILE* fp1 = fopen("result/addthread.txt","w"); 

    ///////////////////////////////////////////////////////
    /*compute*/
    while(P0.source<P0.numVertex){
        P0.target=P0.source+1;
        P1.source=P0.source;
        while(P0.target<P0.numVertex){
            P1.target=P0.target+1;
            if(P1.target>=P0.numVertex){
                compute(h_dest,h_edges,P0,fp1);
            }

            else{
                compute(h_dest,h_edges,P0,fp1);
                compute(h_dest,h_edges,P1,fp1);
            }
            P0.target+=2;

            for (int i=0;i<2*P0.numVertex;i++) {
                P0.h_visited[i] =0;
                P1.h_visited[i] =0;
                }
        } 

    P0.source++;}
    fclose(fp1);

    ///////////////////////////////////////////////////////
    /*free*/
    CUDA_CHECK(hipFree(devmem));
    CUDA_CHECK(hipFree(devmem1));
    free(h_data);
    free(P0.h_label);
    free(P0.h_returned);
    free(P1.h_label);
    free(P1.h_returned);
    hipDeviceReset();
    return 0;
} 