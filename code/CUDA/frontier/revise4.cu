#include "hip/hip_runtime.h"
/*
add POOL
merge memcpy
[X] bfsmain으로 loop end check를 kernel로

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <list>
#include <common/common.h>
#include <common/common_string.h> 

#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 512

struct pool{
    int source; int target; int *d_label; int *d_visited;
    int* d_edges; int* d_dest; int* h_label; int* h_visited; 
    int numVertex; int* h_returned; int *d_frontier;
    int *d_c_frontier_tail; int *d_p_frontier_tail;
    //hipStream_t stream0; hipStream_t stream1;
};

void BFS_host(pool P); 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited); 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail); 

int degree(int* dest,int* edges,int source); 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state, int elen); 

int main(){
    pool P;
    //CUDA_CHECK( hipStreamCreate(&P.stream0) );
    //CUDA_CHECK( hipStreamCreate(&P.stream1) );
    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 
    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen); 

    int *h_data = (int*)malloc((elen+dlen)*sizeof(int)); 
    h_edges= h_data;
    h_dest= &h_data[elen];
    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        //printf("%d\n",i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);
    P.numVertex=elen-1;
    P.h_label=(int*)malloc((4*(P.numVertex)+1)*sizeof(int)); 

    //P.h_visited=(int*)malloc((2*(P.numVertex)+1)*sizeof(int)); 
    P.h_visited=&(P.h_label[2*P.numVertex]);
    int* devmem;
    
    CUDA_CHECK(hipMalloc((void**)&devmem, (elen+dlen+6*P.numVertex+2)*sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&P.d_edges, (elen)*sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&P.d_dest, dlen*sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&P.d_label, 2*P.numVertex*sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&P.d_visited, 2*P.numVertex*sizeof(int)));

    // CUDA_CHECK(hipMalloc((void**)&P.d_frontier, 2*P.numVertex*sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&P.d_c_frontier_tail, sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&P.d_p_frontier_tail, sizeof(int))); 
    P.d_edges=devmem;
    P.d_dest=&(devmem[elen]);
    P.d_label=&(devmem[dlen+elen]);
    P.d_visited=&(devmem[dlen+elen+2*P.numVertex]);

    P.d_frontier=&(devmem[dlen+elen+4*P.numVertex+1]);
    P.d_c_frontier_tail=&(devmem[dlen+elen+6*P.numVertex+1]);
    P.d_p_frontier_tail=&(devmem[dlen+elen+4*P.numVertex]);

    CUDA_CHECK(hipMemcpy(P.d_edges, h_edges, (dlen+elen)*sizeof(int), hipMemcpyHostToDevice)); 

    int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 
    FILE* fp1 = fopen("result/original.txt","w"); 
    while(P.source<P.numVertex){
    P.target=P.source+1;
    while(P.target<P.numVertex){
    int count=0; 
    int N = degree(h_dest,h_edges,P.source);
    int* Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    int check=0;
    for(int j = 0; j<N;j++){
        if(Ad[j]==P.target){
            check=1; 

            break;
        }
    }
    if(check==1){
        P.h_visited[(P.source<<1)+1]=-1;
        P.h_visited[(P.target<<1)]=-1;
        count++;
    }
    free(Ad);
    while(1){ 

        BFS_host(P); 

        std::list<int> list1;
        int num = *P.h_returned;
        if(num==-1) {
            break;
            }
        count++;
        std::list<int>::iterator begin_iter = list1.begin();
        std::list<int>::iterator end_iter = list1.end();
        list1.insert(end_iter, num);
        begin_iter--; 

        int i=num; 

        for (int KK=0; KK<P.numVertex;KK++){
                exclude_S[KK] = P.h_label[KK*2];
                exclude_T[KK] = P.h_label[KK*2+1];
            } 

        while((i>-1)){
            if(i!=num) {
            list1.insert(end_iter, i);}
            i = path(exclude_T,i,h_dest,h_edges,0,elen-1);
            } 

        i=num;
        while((i>-1)){
            if(i!=num) {
            list1.insert(begin_iter, i);
            begin_iter--;}
            i = path(exclude_S,i,h_dest,h_edges,1,elen-1); 

            }
        while (list1.empty()==0) {
            i=list1.front();
            P.h_visited[i<<1]=-1;
            P.h_visited[(i<<1)+1]=-1;
            list1.pop_front(); 
            //if(P.source==0&&P.target==1){printf("%d ",i);}
        } 
        //if(P.source==0&&P.target==1)printf("\n");
        list1.clear();
        //if(P.source==10&&P.target==12) printf(" [%d]\n",count);
        //무한 while
        //check문
    }

    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    P.target++;

    for (int i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
    } 

    P.source++;}
    fclose(fp1); 

    //free

    CUDA_CHECK(hipFree(devmem));
    free(h_data);
    free(P.h_label);
    free(P.h_returned);
    free(exclude_T);
    free(exclude_S);
    hipDeviceReset();
    return 0;
} 

void BFS_host(pool P)
{
    int* h_p_frontier_tail=&P.h_visited[2*P.numVertex];
    *h_p_frontier_tail = 2;

    int S = P.source<<1;
    int T = (P.target<<1)+1; 

    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE>>>(P.d_label, P.d_visited, P.d_frontier, S, T, P.numVertex,P.d_p_frontier_tail);
    for (int i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpy(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice)); 

    int *d_c_frontier = &P.d_frontier[0];
    int *d_p_frontier = &P.d_frontier[P.numVertex];
    int check=0; 

    while (*h_p_frontier_tail > 0&&check==0) { 

        num_blocks = (*h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
        BFS_Bqueue_kernel<<<num_blocks, BLOCK_SIZE>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        
        //CUDA_CHECK(hipMemcpy(P.h_visited, P.d_visited, (2*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, (4*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost)); 

        for(int k=0;k<P.numVertex;k++){
            if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&k!=P.source&&k!=P.target){
                check=1;
                break;
            }
        } 

        int* temp = d_c_frontier;
        d_c_frontier = d_p_frontier;
        d_p_frontier = temp;


    }
    //CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, 2*P.numVertex*sizeof(int), hipMemcpyDeviceToHost)); 

    int min = P.numVertex;
    int meet = -1; 

    for(int k=0;k<P.numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }
    if(P.source==0 && P.target==282) printf("meet : %d, visited : [%d,%d], label : [%d, %d]\n", meet, P.h_visited[2*meet], P.h_visited[2*meet+1], P.h_label[2*meet], P.h_label[2*meet+1]);
    *P.h_returned=meet;
    *P.h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    //__syncthreads();
        if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }

}
