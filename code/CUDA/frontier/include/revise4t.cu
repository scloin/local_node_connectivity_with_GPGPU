#include "hip/hip_runtime.h"
#include <revise4t.hpp>
#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 512

void BFS_host(pool P)
{
    int* h_p_frontier_tail=&P.h_visited[2*P.numVertex];
    *h_p_frontier_tail = 2;

    int S = P.source<<1;
    int T = (P.target<<1)+1; 
    int i; int k; int * temp;
    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(P.d_label, P.d_visited, P.d_frontier, S, T, P.numVertex,P.d_p_frontier_tail);
    for (i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpyAsync(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice,P.stream)); 

    int *d_c_frontier = &P.d_frontier[0];
    int *d_p_frontier = &P.d_frontier[P.numVertex];
    int check=0; 
    
    while (*h_p_frontier_tail > 0&&check==0) { 

        num_blocks = (*h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
        // BFS_Bqueue_kernel<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        BFS_Bqueue<<<num_blocks, BLOCK_SIZE,0,P.stream>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        
        //CUDA_CHECK(hipMemcpy(P.h_visited, P.d_visited, (2*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpyAsync(P.h_label, P.d_label, (4*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost,P.stream)); 

        for(k=0;k<P.numVertex;k++){
            if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&k!=P.source&&k!=P.target){
                check=1;
                break;
            }
        } 
        temp = d_c_frontier;
        d_c_frontier = d_p_frontier;
        d_p_frontier = temp;


    }
    //CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, 2*P.numVertex*sizeof(int), hipMemcpyDeviceToHost)); 

    int min = P.numVertex;
    int meet = -1; 
    //printf("\n...\n");
    for(k=0;k<P.numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1&&P.h_label[2*k]*P.h_label[2*k+1]!=0){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }
    //if(P.source==0 && P.target==282) printf("meet : %d, visited : [%d,%d], label : [%d, %d]\n", meet, P.h_visited[2*meet], P.h_visited[2*meet+1], P.h_label[2*meet], P.h_label[2*meet+1]);
    *P.h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 
    __shared__ int w_queue[16][16], our_c_frontier_s_tail[16];
    __shared__ int w_tail[16];

    if (threadIdx.x <16) {
        c_frontier_tail_s = 0;
        w_tail[threadIdx.x]=0;
    }
    __syncthreads(); 
    const int wid = threadIdx.x%16;
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (was_visited==0) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                //const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                const int my_w_tail = atomicAdd(&w_tail[wid], 1);
                if (my_w_tail < 16) {
                    w_queue[wid][my_w_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    w_tail[wid]=16;
                    const int my_block_tail = atomicAdd(&c_frontier_tail_s, 1);
                    if (my_block_tail < BLOCK_QUEUE_SIZE) {
                        c_frontier_s[my_block_tail] = (dest[i]<<1)+my_state;
                    }
                    else {
                        c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                        const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                        c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                    }
                    //c_frontier_s[my_block_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x <16) {
        our_c_frontier_s_tail[wid] = atomicAdd(&c_frontier_tail_s, w_tail[wid]);
        //printf("our_c_- %d, w_tail%d\n", our_c_frontier_s_tail[wid], w_tail[wid]);
    }
    __syncthreads(); 

    for (int i = threadIdx.x/16; i < w_tail[wid]; i += 16) {
        //printf("tid=%d, wqueue[%d][%d]=%d, c_frontier_s[%d]\n", threadIdx.x,wid,i,w_queue[wid][i],our_c_frontier_s_tail[wid] + i);
        if(our_c_frontier_s_tail[wid] + i>=BLOCK_QUEUE_SIZE) {
            const int second_global_tail = atomicAdd(c_frontier_tail, 1);
            c_frontier[second_global_tail] = w_queue[wid][i];
            }
        else{
            c_frontier_s[our_c_frontier_s_tail[wid] + i] = w_queue[wid][i];
            }    
        }
    __syncthreads(); 


    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    __syncthreads();
        if (tid == 0) {

        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }

}

pool init_pool(int elen, int dlen, int* devmem, int* d_edges){

    pool P;
    CUDA_CHECK( hipStreamCreate(&P.stream) );
    P.numVertex=elen-1;

    P.d_edges=d_edges;
    P.d_dest=&d_edges[elen];

    P.h_label=(int*)malloc((4*(P.numVertex)+2)*sizeof(int)); 
    //CUDA_CHECK(hipHostMalloc((void**)&P.h_label, (4*(P.numVertex)+2)*sizeof(int)));
    P.h_visited=&(P.h_label[2*P.numVertex]);

    P.d_label=devmem;
    P.d_visited=&(devmem[2*P.numVertex]);

    P.d_frontier=&(devmem[4*P.numVertex+1]);
    P.d_c_frontier_tail=&(devmem[6*P.numVertex+1]);
    P.d_p_frontier_tail=&(devmem[4*P.numVertex]);

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 

    return P;
}

void compute(int* h_dest,int * h_edges, pool P,FILE* fp1){
    int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 
    int init = P.target; 
    int count; int N; int* Ad; int num; int i; int j; int KK;
    while(P.source<P.numVertex){
        P.target=P.source+init;
        while(P.target<P.numVertex){
    count=0; 
    //verify that it is connected directly
    N = degree(h_dest,h_edges,P.source);
    Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    //int check=0;
    for(j = 0; j<N;j++){
        if(Ad[j]==P.target){
            P.h_visited[(P.source<<1)+1]=-1;
            P.h_visited[(P.target<<1)]=-1;
            count++; 
            break;
        }
    }
    free(Ad);
    //loop while count every component
    while(1){ 
    
        //BFS_host(P);
        BFS_host(P);
        num = *P.h_returned;
        if(num==-1) {
            break;
            }
        count++;

        i=num;
        j=num; 
        P.h_visited[i<<1]=-1;
        P.h_visited[(i<<1)+1]=-1;

        for (KK=0; KK<P.numVertex;KK++){
                exclude_S[KK] = P.h_label[KK*2];
                exclude_T[KK] = P.h_label[KK*2+1];
            } 

        int tempi=0;
        int tempj=0;
        int error =0;
        while((i>-1)||(j>-1)){
            if(i>-1){
                if(i!=num) {
                P.h_visited[i<<1]=-1;
                P.h_visited[(i<<1)+1]=-1;
                }
                tempi=i;
                i = path(exclude_T,i,h_dest,h_edges,0,P.numVertex);
                if(tempi==i){
                    error=1;
                    break;
                }
            }
            if(j>-1){
                if(j!=num) {
                P.h_visited[j<<1]=-1;
                P.h_visited[(j<<1)+1]=-1;
                }
                tempj=j;
                j = path(exclude_S,j,h_dest,h_edges,1,P.numVertex);
                if(tempj==j){
                    error=1;
                    break;
                }
            }
        }
        if(error==1){
            count--;
            break;
        }
    }
    //record count
    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    
    P.target+=4;
    //P.target++;
    for (i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
        }
    P.source++;
    }

    free(exclude_S);
    free(exclude_T);
}

__global__ void BFS_Bqueue(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    //__syncthreads();
        if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 
