#include "hip/hip_runtime.h"
/*
add POOL
merge memcpy
[X] bfsmain으로 loop end check를 kernel로
loopend 를 kernel로
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <list>
#include <common/common.h>
#include <common/common_string.h> 

#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 32 

struct pool{
    int source; int target; int *d_label; int *d_visited;
    int* d_edges; int* d_dest; int* h_label; int* h_visited; 
    int numVertex; int* h_returned; int *d_frontier;
    int *d_c_frontier_tail; int *d_p_frontier_tail;
    int*check; int*h_check;
    //hipStream_t stream0; hipStream_t stream1;
};

void BFS_host(pool P); 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited); 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail, int* check); 

__global__ void checking(int num,int* h_visited, int source, int target, int* check,int *d_c_frontier,int*d_p_frontier);

int degree(int* dest,int* edges,int source); 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state, int elen); 

int main(){
    pool P;
    //CUDA_CHECK( hipStreamCreate(&P.stream0) );
    //CUDA_CHECK( hipStreamCreate(&P.stream1) );
    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 
    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen); 

    int *h_data = (int*)malloc((elen+dlen)*sizeof(int)); 
    h_edges= h_data;
    h_dest= &h_data[elen];
    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        //printf("%d\n",i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);
    P.numVertex=elen-1;
    P.h_label=(int*)malloc((4*(P.numVertex)+2)*sizeof(int)); 

    //P.h_visited=(int*)malloc((2*(P.numVertex)+1)*sizeof(int)); 
    P.h_visited=&(P.h_label[2*P.numVertex]);
    //P.h_p_frontier_tail=&(P.h_label[4*P.numVertex]);
    P.h_check  =&(P.h_label[4*P.numVertex+1]);
    int* devmem;
    
    CUDA_CHECK(hipMalloc((void**)&devmem, (elen+dlen+6*P.numVertex+3)*sizeof(int)));

    P.d_edges=devmem;
    P.d_dest=&(devmem[elen]);
    P.d_label=&(devmem[dlen+elen]);
    P.d_visited=&(P.d_label[2*P.numVertex]);

    P.d_p_frontier_tail=&(P.d_label[4*P.numVertex]);
    P.check=&(P.d_label[4*P.numVertex+1]);

    P.d_frontier=&(P.d_label[4*P.numVertex+2]);
    P.d_c_frontier_tail=&(P.d_label[6*P.numVertex+2]);

    CUDA_CHECK(hipMemcpy(P.d_edges, h_edges, (dlen+elen)*sizeof(int), hipMemcpyHostToDevice)); 

    //P.h_check = (int*)malloc(sizeof(int)); 

    int *exclude_S=(int *)malloc(P.numVertex*sizeof(int));
    int *exclude_T=(int *)malloc(P.numVertex*sizeof(int)); 

    P.h_returned=(int*)malloc(sizeof(int));
    P.source=0;
    P.target=1; 
    FILE* fp1 = fopen("result/test3.txt","w"); 

    while(P.source<P.numVertex){
        P.target=P.source+1;

    while(P.target<P.numVertex){
    int count=0; 
    int N = degree(h_dest,h_edges,P.source);
    int* Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[P.source]]),N*sizeof(int)); 

    int check=0;
    for(int j = 0; j<N;j++){
        if(Ad[j]==P.target){
            check=1; 

            break;
        }
    }
    if(check==1){
        P.h_visited[(P.source<<1)+1]=-1;
        P.h_visited[(P.target<<1)]=-1;
        count++;
    }
    free(Ad);
    while(1){ 

    BFS_host(P); 

    std::list<int> list1;
    int num = *P.h_returned;
    if(num==-1) {
        break;
        }
    count++;
    std::list<int>::iterator begin_iter = list1.begin();
    std::list<int>::iterator end_iter = list1.end();
    list1.insert(end_iter, num);
    begin_iter--; 

    int i=num; 

    for (int KK=0; KK<P.numVertex;KK++){
            exclude_S[KK] = P.h_label[KK*2];
            exclude_T[KK] = P.h_label[KK*2+1];
        } 

    while((i>-1)){
        if(i!=num) {
        list1.insert(end_iter, i);}
        i = path(exclude_T,i,h_dest,h_edges,0,elen-1);
        } 
       
    i=num;
    while((i>-1)){
        if(i!=num) {
        list1.insert(begin_iter, i);
        begin_iter--;}
        i = path(exclude_S,i,h_dest,h_edges,1,elen-1);
        }
    while (list1.empty()==0) {
        i=list1.front();
        P.h_visited[i<<1]=-1;
        P.h_visited[(i<<1)+1]=-1;
        list1.pop_front(); 
        //if(P.source==10&&P.target==12){printf("%d ",i);}
    } 
    
    list1.clear();
    //if(P.source==10&&P.target==12) printf(" [%d]\n",count);
    //무한 while
    //check문
    }

    fprintf(fp1,"[%d, %d] %d\n", P.source, P.target, count);
    P.target++;

    // for (int i=0; i<2*P.numVertex;i++)
    //     printf("%2d ", h_label[i]);
    // printf("\n");

    for (int i=0;i<2*P.numVertex;i++) {
        P.h_visited[i] =0;
        }
    } 

    P.source++;}
    fclose(fp1); 

    //free
    // CUDA_CHECK(hipFree(P.d_edges));
    // CUDA_CHECK(hipFree(P.d_dest));
    // CUDA_CHECK(hipFree(P.d_label));
    // CUDA_CHECK(hipFree(P.d_visited)); 

    // CUDA_CHECK(hipFree(P.d_frontier));
    // CUDA_CHECK(hipFree(P.d_c_frontier_tail));
    // CUDA_CHECK(hipFree(P.d_p_frontier_tail));
    CUDA_CHECK(hipFree(devmem));
    free(h_data);
    //free(P.h_check);
    //free(h_dest);
    //free(h_edges);
    free(P.h_label);
    //free(P.h_visited);
    free(P.h_returned);
    free(exclude_T);
    free(exclude_S);
    //CUDA_CHECK( hipStreamDestroy(P.stream0));
    //CUDA_CHECK( hipStreamDestroy(P.stream1));
    hipDeviceReset();
    return 0;
} 

void BFS_host(pool P)
{
    int* h_p_frontier_tail=&P.h_visited[2*P.numVertex];
    *h_p_frontier_tail = 2;

    int S = P.source<<1;
    int T = (P.target<<1)+1; 

    int num_blocks = ((2*P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE>>>(P.d_label, P.d_visited, P.d_frontier, S, T, P.numVertex,P.d_p_frontier_tail, P.check);
    for (int i=0;i<2*P.numVertex;i++) {
        if(P.h_visited[i]!=-1)
            P.h_visited[i] =0;
        }
    P.h_visited[S] =1;P.h_visited[T] =1; 
    CUDA_CHECK(hipMemcpy(P.d_visited, P.h_visited, 2*P.numVertex*sizeof(int), hipMemcpyHostToDevice)); 

    int *d_c_frontier = &P.d_frontier[0];
    int *d_p_frontier = &P.d_frontier[P.numVertex];
    *P.h_check=0; 
    int * temp;

    while (*h_p_frontier_tail > 0&&*P.h_check==0) { 

        num_blocks = (*h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
        int num_blocks1 = ((P.numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
        BFS_Bqueue_kernel<<<num_blocks, BLOCK_SIZE>>>(d_p_frontier, P.d_p_frontier_tail,d_c_frontier, P.d_c_frontier_tail, P.d_edges, P.d_dest, P.d_label, P.d_visited);
        
        checking<<<num_blocks1, BLOCK_SIZE>>>(P.numVertex, P.d_visited, P.source, P.target, P.check, d_c_frontier ,d_p_frontier);
        
        //hipDeviceSynchronize();
        //CUDA_CHECK(hipMemcpy(P.h_check, P.check, sizeof(int), hipMemcpyDeviceToHost)); 
        //CUDA_CHECK(hipMemcpy(h_p_frontier_tail, P.d_p_frontier_tail, 2*sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, (4*P.numVertex+2)*sizeof(int), hipMemcpyDeviceToHost)); 

        temp = d_c_frontier;
        d_c_frontier = d_p_frontier;
        d_p_frontier = temp;
    }
    //CUDA_CHECK(hipMemcpy(P.h_label, P.d_label, (4*P.numVertex+1)*sizeof(int), hipMemcpyDeviceToHost)); 

    int min = P.numVertex;
    int meet = -1; 

    for(int k=0;k<P.numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(P.h_visited[2*k]==1&&P.h_visited[2*k+1]==1){
            if(min>P.h_label[2*k]+P.h_label[2*k+1]){
                min = P.h_label[2*k]+P.h_label[2*k+1];
                meet =k;
            }
        }
    }
    *P.h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
    //__syncthreads();
        if (tid == 0) {
        *p_frontier_tail = atomicExch(c_frontier_tail, 0);
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail, int * check){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }
    *check=0;

}

__global__ void checking(int num,int* h_visited, int source, int target, int* check,int *d_c_frontier,int*d_p_frontier){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid<num)
        if(h_visited[2*tid]==1&&h_visited[2*tid+1]==1&&tid!=source&&tid!=target){
            *check=1;
        }
    // if(tid==0){
    //     int *temp = d_c_frontier;
    //     d_c_frontier = d_p_frontier;
    //     d_p_frontier = temp;}
}