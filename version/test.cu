#include "hip/hip_runtime.h"
//nvcc -o test test/cu -I.. -arch=sm_35 -rdc=true
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <list>
#include <common/common.h>
#include <common/common_string.h> 

#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 32 

__global__ void add(int* A, int* B, int* C);
//__global__ void filter(int* A, int* C);
//__global__ void mainstream(int* A, int* B, int* C);

__global__ void BFSmain(int* in,int* out, int* edges, int*dest, int N,int* kk);
__global__ void matmul(int* input, int* output, int* edges, int*dest, int NUM, int T);
__global__ void filter(int* input, int* output, int NUM);
__global__ void checking(int* input, int* input1, int* kk, int NUM);

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen);
int degree(int* dest,int* edges,int source);

int main(){
    //init&alloc
        // int A[]={0,1,2,3,4,5,6,7};
        // int B[]={0,1,2,3,4,5,6,7};
        // //int C[]={0,0,0,0,0,0,0,0};
        // int* C=(int*)calloc(8,sizeof(int));
        // int* Ac, *Bc, *Cc;
        // CUDA_CHECK(hipMalloc((void**)&Ac, 8*sizeof(int))); 
        // CUDA_CHECK(hipMalloc((void**)&Bc, 8*sizeof(int)));
        // CUDA_CHECK(hipMalloc((void**)&Cc, 8*sizeof(int)));
        
        // CUDA_CHECK(hipMemcpy(Ac, A, 8*sizeof(int), hipMemcpyHostToDevice)); 
        // CUDA_CHECK(hipMemcpy(Bc, B, 8*sizeof(int), hipMemcpyHostToDevice)); 
        // CUDA_CHECK(hipMemcpy(Cc, C, 8*sizeof(int), hipMemcpyHostToDevice)); 

    //read file&gengraph
        FILE* fp = fopen("./graph/graph.txt","r");
        int dlen; int elen; 
        int * h_dest; int * h_edges; 
        fscanf(fp, "%d ", &dlen);
        fscanf(fp, "%d ", &elen); 
        h_dest = (int*)malloc(dlen*sizeof(int));
        h_edges = (int*)malloc(elen*sizeof(int)); 
        int k =0;
        int i;
        while(k<dlen){
            fscanf(fp, "%d ", &i);
            h_dest[k]=i;
            k++;
        }
        k=0;
        while(k<elen){
            fscanf(fp, "%d ", &i);
            //printf("%d\n",i);
            h_edges[k]=i;
            k++;
        }
        fclose(fp);
        int *d_edges, *d_dest;
        CUDA_CHECK(hipMalloc((void**)&d_edges, elen*sizeof(int)));
        CUDA_CHECK(hipMalloc((void**)&d_dest, dlen*sizeof(int)));
        CUDA_CHECK(hipMemcpy(d_edges, h_edges, elen*sizeof(int), hipMemcpyHostToDevice)); 
        CUDA_CHECK(hipMemcpy(d_dest, h_dest, dlen*sizeof(int), hipMemcpyHostToDevice)); 
    int num=elen-1;
    int* h_in, *d_in, *h_out, *d_out;
    int* kk;
    int source; int target; int count;
    h_in=(int*)calloc(2*num,sizeof(int));
    h_out=(int*)calloc(2*num,sizeof(int));
    CUDA_CHECK(hipMalloc((void**)&d_in, 2*num*sizeof(int))); 
    CUDA_CHECK(hipMalloc((void**)&d_out, 2*num*sizeof(int))); 
    CUDA_CHECK(hipMalloc((void**)&kk, 2*sizeof(int)));

    FILE* fp1 = fopen("result/test4.txt","w");

    source=0; target=12; 
    while(source<num){
    target=source+1;
    while(target<num){
    count=0; int meet = -1;

    int exclude_S[num];
    int exclude_T[num];
    do{
    h_in[source]=1; h_in[target+num]=1; h_out[source]=-1; h_out[target+num]=-1;
    CUDA_CHECK(hipMemset(kk, 0, 2*sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_in, h_in, 2*num*sizeof(int), hipMemcpyHostToDevice)); 
    CUDA_CHECK(hipMemcpy(d_out, h_out, 2*num*sizeof(int), hipMemcpyHostToDevice)); 

    //while()
    BFSmain<<<1,1>>>(d_in, d_out, d_edges, d_dest, num,kk);
    CUDA_CHECK(hipMemcpy(h_out, d_out, 2*num*sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_in, d_in, 2*num*sizeof(int), hipMemcpyDeviceToHost));
    int min = num;
    meet = -1; 

    for(int k=0;k<num;k++){
        if(h_in[k]==1&&h_in[k+num]==1){
            if((min>h_out[k]+h_out[k+num])&&(h_out[k]+h_out[k+num]>=0)){

                if((k==source || k==target)&&(h_out[k]+h_out[k+num]==0)){
                    count+=1;
                    h_in[source]=-1;
                    h_in[source+num]=-1;
                    h_in[target]=-1;
                    h_in[target+num]=-1;
                    continue;
                }
                else if((k==source || k==target)){
                    h_in[source]=-1;
                    h_in[source+num]=-1;
                    h_in[target]=-1;
                    h_in[target+num]=-1;
                    continue;
                }
                min = h_out[k]+h_out[k+num];
                meet =k;
            }
        }}
    for(int k=0;k<num;k++){
        if(h_out[k]==0) h_out[k]=-1;
        else if(h_out[k]==-1) h_out[k]=0;
        if(h_out[k+num]==0) h_out[k+num]=-1;
        else if(h_out[k+num]==-1) h_out[k+num]=0;
    }
    
    if(meet==-1){break;}
    else{
    count++;
    std::list<int> list1;    
    std::list<int>::iterator begin_iter = list1.begin();
    std::list<int>::iterator end_iter = list1.end();
    list1.insert(end_iter, meet);
    
    int i=meet; 
    //printf("%d\n",i);
    for (int KK=0; KK<num;KK++){
            exclude_S[KK] = h_out[KK];
            exclude_T[KK] = h_out[KK+num];
        } 

    while((i>-1)){
        if(i!=meet) {
        list1.insert(end_iter, i);}
        i = path(exclude_T,i,h_dest,h_edges,0,num);
        } 

    i=meet;
    while((i>-1)){
        if(i!=meet) {
        list1.insert(begin_iter, i);
        begin_iter--;}
        i = path(exclude_S,i,h_dest,h_edges,1,num); 

        }
    
    while (list1.empty()==0) {
        i=list1.front();
        h_in[i]=-1;
        h_in[i+num]=-1;
        list1.pop_front(); 
        //if(source==10&&target==12){printf("%d ",i);}
    } 

    list1.clear();
    //if(source==10&&target==12) printf(" [%d]\n",count);
    }
    for (int i=0;i<2*num;i++) {
        h_out[i]=0;
        if(h_in[i]!=-1) h_in[i] =0;
    }
    }while(meet!=-1);
    fprintf(fp1,"[%d, %d] %d\n", source, target, count);
    target++;
        for (int i=0;i<2*num;i++) {
        h_out[i]=0;
        h_in[i]=0;
        }
    } 
    source++;
    }
    //printf("%d\n", count);
    fclose(fp1);

    free(h_in); free(h_out); free(h_dest); free(h_edges);
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    CUDA_CHECK(hipFree(d_dest));
    CUDA_CHECK(hipFree(d_edges));
    CUDA_CHECK(hipFree(kk));
    return 0;
}

/*
    __global__ void add(int* A,int *B,int *C){
        const int tid = blockIdx.x*blockDim.x + threadIdx.x;
        
        if(tid<8){
            C[tid]=A[tid]*B[tid];
        }__syncthreads();
    }

    __global__ void filter(int* A,int *C){
        const int tid = blockIdx.x*blockDim.x + threadIdx.x;
        if(tid<8){
            C[tid]=C[tid]%9;
            A[tid]=C[tid];
        }__syncthreads();
    }

    __global__ void mainstream(int* A,int *B,int *C){
        int i=0;
        while(i<5){
        add<<<1,8>>>(A,B,C);
        //__syncthreads();
        hipDeviceSynchronize();
        filter<<<8,1>>>(A,C);
        hipDeviceSynchronize();
        i++;
        for(int i=0; i<8;i++) printf("%d ",C[i]);
        printf("\n");
        }
    }
*/
__global__ void BFSmain(int* in,int* out, int* edges, int*dest, int N, int* kk){

    int ccheck=1; int check=0;
    int num_blocks = ((N*N)+BLOCK_SIZE-1) / BLOCK_SIZE;
    int num_blocks1 = (N+BLOCK_SIZE-1) / BLOCK_SIZE;
    int* in1=&in[N]; int* out1=&out[N];
    while(kk[0]==0){
        check=kk[1];
        kk[1]=0;
        matmul<<<num_blocks,BLOCK_SIZE>>>(in,out, edges, dest, N, ccheck);
        matmul<<<num_blocks,BLOCK_SIZE>>>(in1,out1, edges, dest, N, ccheck);
        hipDeviceSynchronize();

        filter<<<num_blocks1,BLOCK_SIZE>>>(in,out, N);
        filter<<<num_blocks1,BLOCK_SIZE>>>(in1,out1, N);
        hipDeviceSynchronize();
        checking<<<num_blocks1,BLOCK_SIZE>>>(in,in1,kk,N);
        hipDeviceSynchronize();
        ccheck++;
        if(check==kk[1]) kk[0]=-1;
    }
}

__global__ void matmul(int* input, int* output, int* edges, int*dest, int NUM,int T){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int N = tid/NUM; int M = tid%NUM;
    int k=0;
    if(M<(edges[N+1]-edges[N])){
        k=input[dest[edges[N]+M]];
        if(output[N]==0&&k!=0&&input[N]!=-1&&k!=-1){
            atomicExch(&output[N],T);
        }
    }
    __syncthreads();
}

__global__ void filter(int* input, int* output, int NUM){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    //int check0=0;
    //int dd[1];
    //dd[0]=1;
    if(tid<NUM){
        //printf("              wht?%d[%d]\n",tid,output[tid]);
        if(output[tid]==0) {
            //printf("                  if %d\n",tid);
            //atomicAdd(&check0,1);
        }
        else if(input[tid]==0){
            input[tid]=1;
            //printf("                  elseif %d\n",tid);
        }
        else if(input[tid]==1){
            //input[tid]=0;
            //printf("                  else %d\n",tid);
            }
    }
    // if(tid==1){
    //     for(int i=0; i<NUM;i++) printf("%d ",input[i]);
    //         printf("\n");}
    __syncthreads();
}

__global__ void checking(int* input, int* input1, int* kk, int NUM){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid<NUM){
        if(input[tid]==input1[tid]&&input1[tid]==1){
            kk[0]=tid;
        }
        else if(input[tid]==0||input1[tid]==0){
            atomicAdd(&kk[1],input[tid]+input1[tid]);

        }
    }
    __syncthreads();
}

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 
    

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 
