#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <list>
#include <common/common.h>
#include <common/common_string.h> 

#define BLOCK_SIZE 512
#define BLOCK_QUEUE_SIZE 32 

void BFS_host(int source, int target, int *d_label, int *d_visited, int* d_edges, int* d_dest, int* h_label,int* h_visited, int numVertex, int* h_returned, int *d_frontier,int *d_c_frontier_tail,int *d_p_frontier_tail ,hipStream_t stream0 ,hipStream_t stream1); 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited); 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail); 

int degree(int* dest,int* edges,int source); 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state, int elen); 

int main(){
    hipStream_t stream0; CUDA_CHECK( hipStreamCreate(&stream0) );
    hipStream_t stream1; CUDA_CHECK( hipStreamCreate(&stream1) );
    FILE* fp = fopen("./graph/graph.txt","r");
    int dlen;
    int elen; 

    int * h_dest;
    int * h_edges; 

    fscanf(fp, "%d ", &dlen);
    fscanf(fp, "%d ", &elen); 

    h_dest = (int*)malloc(dlen*sizeof(int));
    h_edges = (int*)malloc(elen*sizeof(int)); 

    int k =0;
    int i;
    while(k<dlen){
        fscanf(fp, "%d ", &i);
        h_dest[k]=i;
        k++;
    }
    k=0;
    while(k<elen){
        fscanf(fp, "%d ", &i);
        //printf("%d\n",i);
        h_edges[k]=i;
        k++;
    }
    fclose(fp);
    int *h_label;
    h_label=(int*)malloc(2*(elen-1)*sizeof(int)); 

    int *h_visited;
    h_visited=(int*)malloc(2*(elen-1)*sizeof(int)); 

    int *d_edges, *d_dest, *d_label, *d_visited;
    CUDA_CHECK(hipMalloc((void**)&d_edges, (elen)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_dest, dlen*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_label, 2*(elen-1)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_visited, 2*(elen-1)*sizeof(int)));


    int *d_frontier, *d_c_frontier_tail, *d_p_frontier_tail;
    CUDA_CHECK(hipMalloc((void**)&d_frontier, 2*(elen-1)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_c_frontier_tail, sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_p_frontier_tail, sizeof(int))); 

    CUDA_CHECK(hipMemcpyAsync(d_edges, h_edges, (elen)*sizeof(int), hipMemcpyHostToDevice,stream0)); 

    CUDA_CHECK(hipMemcpyAsync(d_dest, h_dest, dlen*sizeof(int), hipMemcpyHostToDevice,stream1)); 

    int *exclude_S=(int *)malloc((elen-1)*sizeof(int));
    int *exclude_T=(int *)malloc((elen-1)*sizeof(int)); 

    int * h_returned=(int*)malloc(sizeof(int));
    int source=0;
    int target=0; 

    FILE* fp1 = fopen("result/test2.txt","w"); 

    while(source<(elen-1)){
    target=source+1;
    while(target<(elen-1)){
    int count=0; 
    int N = degree(h_dest,h_edges,source);
    int* Ad=(int*)malloc(N*sizeof(int));
    memcpy(Ad,&(h_dest[h_edges[source]]),N*sizeof(int)); 

    int check=0;
    for(int j = 0; j<N;j++){
        if(Ad[j]==target){
            check=1; 

            break;
        }
    }
    if(check==1){
        h_visited[(source<<1)+1]=-1;
        h_visited[(target<<1)]=-1;
        count++;
    }
    free(Ad);
    if(1) {
    

    while(1){ 

    BFS_host(source, target, d_label, d_visited, d_edges, d_dest, h_label, h_visited, (elen-1), h_returned, d_frontier, d_c_frontier_tail, d_p_frontier_tail,stream0,stream1); 

    std::list<int> list1;
    int num = *h_returned;
    if(num==-1) {
        break;
        }
    count++;
    std::list<int>::iterator begin_iter = list1.begin();
    std::list<int>::iterator end_iter = list1.end();
    list1.insert(end_iter, num);
    begin_iter--; 

    int i=num; 

    for (int KK=0; KK<(elen-1);KK++){
            exclude_S[KK] = h_label[KK*2];
            exclude_T[KK] = h_label[KK*2+1];
        } 

    while((i>-1)){
        if(i!=num) {
        list1.insert(end_iter, i);}
        i = path(exclude_T,i,h_dest,h_edges,0,elen-1);
        } 

    i=num;
        for (int i=0; i<(elen-1);i++)
            printf("%2d ", exclude_S[i]);
    printf("\n");
    while((i>-1)){
        if(i!=num) {
        list1.insert(begin_iter, i);
        begin_iter--;}
        i = path(exclude_S,i,h_dest,h_edges,1,elen-1); 

        }
    while (list1.empty()==0) {
        i=list1.front();
        h_visited[i<<1]=-1;
        h_visited[(i<<1)+1]=-1;
        list1.pop_front(); 

    } 

    list1.clear();
    }//무한 while
    //check문
    }

    fprintf(fp1,"[%d, %d] %d\n", source, target, count);
    target++;

    // for (int i=0; i<2*(elen-1);i++)
    //     printf("%2d ", h_label[i]);
    // printf("\n");

    for (int i=0;i<2*(elen-1);i++) {
        h_visited[i] =0;
        }
    } 

    source++;}
    fclose(fp1); 

    //free
    CUDA_CHECK(hipFree(d_edges));
    CUDA_CHECK(hipFree(d_dest));
    CUDA_CHECK(hipFree(d_label));
    CUDA_CHECK(hipFree(d_visited)); 

    CUDA_CHECK(hipFree(d_frontier));
    CUDA_CHECK(hipFree(d_c_frontier_tail));
    CUDA_CHECK(hipFree(d_p_frontier_tail)); 

    free(h_dest);
    free(h_edges);
    hipHostFree(h_label);
    hipHostFree(h_visited);
    free(h_returned);
    free(exclude_T);
    free(exclude_S);
    CUDA_CHECK( hipStreamDestroy(stream0));
    CUDA_CHECK( hipStreamDestroy(stream1));
    hipDeviceReset();
    return 0;
} 

void BFS_host(int source, int target, int *d_label, int *d_visited, int* d_edges, int* d_dest, int* h_label,int* h_visited, int numVertex, int* h_returned, int *d_frontier,int *d_c_frontier_tail,int *d_p_frontier_tail,hipStream_t stream0 ,hipStream_t stream1)
{
    int h_p_frontier_tail = 2; 

    int S = source<<1;
    int T = (target<<1)+1; 

    int num_blocks = ((2*numVertex)+BLOCK_SIZE-1) / BLOCK_SIZE;
    memset_kernel<<<num_blocks, BLOCK_SIZE, 0, stream0>>>(d_label, d_visited, d_frontier, S, T, numVertex,d_p_frontier_tail);
    for (int i=0;i<2*numVertex;i++) {
        if(h_visited[i]!=-1)
            h_visited[i] =0;
        }
    h_visited[S] =1;h_visited[T] =1; 
    CUDA_CHECK(hipMemcpyAsync(d_visited, h_visited, 2*numVertex*sizeof(int), hipMemcpyHostToDevice,stream1)); 

    int *d_c_frontier = &d_frontier[0];
    int *d_p_frontier = &d_frontier[numVertex];
    int check=0; 

    while (h_p_frontier_tail > 0&&check==0) { 

        num_blocks = (h_p_frontier_tail+BLOCK_SIZE-1) / BLOCK_SIZE;
        BFS_Bqueue_kernel<<<num_blocks, BLOCK_SIZE>>>(d_p_frontier, d_p_frontier_tail, d_c_frontier, d_c_frontier_tail, d_edges, d_dest, d_label, d_visited);
        CUDA_CHECK(hipMemcpyAsync(&h_p_frontier_tail, d_c_frontier_tail, sizeof(int), hipMemcpyDeviceToHost,stream0));
        CUDA_CHECK(hipMemcpyAsync(d_p_frontier_tail, d_c_frontier_tail, sizeof(int), hipMemcpyDeviceToDevice,stream1));
        CUDA_CHECK(hipMemcpyAsync(h_visited, d_visited, 2*numVertex*sizeof(int), hipMemcpyDeviceToHost,stream0));
        CUDA_CHECK(hipMemsetAsync(d_c_frontier_tail, 0, sizeof(int),stream1)); 

        for(int k=0;k<numVertex;k++){
            if(h_visited[2*k]==1&&h_visited[2*k+1]==1&&k!=source&&k!=target){
                check=1;
                break;
            }
        } 

        int* temp = d_c_frontier;
        d_c_frontier = d_p_frontier;
        d_p_frontier = temp;


    }
    CUDA_CHECK(hipMemcpyAsync(h_label, d_label, 2*numVertex*sizeof(int), hipMemcpyDeviceToHost, stream0)); 

    int min = numVertex;
    int meet = -1; 

    for(int k=0;k<numVertex;k++){// h_label[2*k+1], h_visited[2*k], h_visited[2*k+1]);
        if(h_visited[2*k]==1&&h_visited[2*k+1]==1){
            if(min>h_label[2*k]+h_label[2*k+1]){
                min = h_label[2*k]+h_label[2*k+1];
                meet =k;
            }
        }
    }
    *h_returned=meet;
} 

__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited){
    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s, our_c_frontier_tail; 

    if (threadIdx.x == 0)
        c_frontier_tail_s = 0;
    __syncthreads(); 

    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid]>>1;
        const int my_state = p_frontier[tid]&1;
        for (int i = edges[my_vertex]; i < edges[my_vertex+1]; i++) {
            int was_visited=2;
            if(visited[(dest[i]<<1)+my_state]!=-1)
                was_visited = atomicExch(&(visited[(dest[i]<<1)+my_state]), 1);
            if (!was_visited) {
                label[(dest[i]<<1)+my_state] = label[p_frontier[tid]] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = (dest[i]<<1)+my_state;
                }
                else {
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = (dest[i]<<1)+my_state;
                }
            }
        }
    }
    __syncthreads(); 

    if (threadIdx.x == 0) {
        our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
    }
    __syncthreads(); 

    for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
        c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
    }
} 

int path(int* exclude, int num, int* h_dest, int* h_edges, int state,int elen){
    int numT=num;
    int check=0;
    int i=0; 

    int N = degree(h_dest,h_edges,num); 

    if(exclude[numT]==1){
        if(state==0) {
            return -2;
        }else {
            return -1;
        }
    }
    else{
    int* AA=(int*)malloc(N*sizeof(int));
    memcpy(AA,&(h_dest[h_edges[num]]),N*sizeof(int));
        for (i = 0; i < elen; i++){
            if (exclude[i] == exclude[numT]-1) {
                for(int j = 0; j<degree(h_dest,h_edges,numT);j++){
                    if(AA[j]==i){
                        check=1;
                        break;
                    }
                }
            }
            if(check==1) break;
        }
    free(AA);
    }
    if(check!=1) i=numT; 

    return i;
} 

int degree(int* dest,int* edges,int source){ 

    return edges[source+1]-edges[source];
} 

__global__ void memset_kernel(int* d_label, int* d_visited, int* d_frontier, int S, int T, int NUM, int* d_p_frontier_tail){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < NUM*2) {
        *(d_label+tid) = -1;
        *(d_frontier+tid) =0;
        if (tid==S||tid==T) {
            *(d_label+tid) = 0;
                }
        if (tid ==NUM) *(d_frontier+tid) = S;
        if (tid ==NUM+1) *(d_frontier+tid) = T;
                *(d_p_frontier_tail) =2;
    }

}
